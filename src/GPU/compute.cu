#include "hip/hip_runtime.h"
#include "compute.hh"


namespace GPU {
    void Matrix::fromGpu(double *gpu_rep, unsigned row, unsigned col, size_t pitch) {
        MatrixXd tmp{row, col};
        double *h_d = (double*) std::malloc(sizeof(double) * col * row);

        hipMemcpy2D(h_d, sizeof(double)*col, gpu_rep, pitch, sizeof(double)*col,
                     row, hipMemcpyDeviceToHost);

        for(unsigned i = 0; i < row; ++i)
            for (unsigned j = 0; j < col; ++j)
                tmp(i,j) = h_d[col*i + j];
        Matrix new_matrix{tmp};
        *this = new_matrix;
        std::free(h_d);
    }

    double *Matrix::toGpu(size_t *pitch) const {
        unsigned r = this->rows();
        unsigned c = this->cols();

        double *d_x;
        hipMallocPitch((void **) &d_x, pitch, sizeof(double) * c, r);

        Matrix tmp {this->transpose()};
        double *h_d = tmp.data();
        hipMemcpy2D(d_x, *pitch, h_d, c*sizeof(double), sizeof(double)*c,
                     r, hipMemcpyHostToDevice);
        return (double*)d_x;
    }
}

void computeDim(unsigned width, unsigned height,
                dim3 *block, dim3 *grid) {
    int devId = 0; // There may be more devices!
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devId);

    // int xMaxBlocks = deviceProp.maxGridSize[0];
    // int yMaxBlocks = deviceProp.maxGridSize[1];

    int xThreads = 32; // deviceProp.maxThreadsDim[0];
    int yThreads = 32; // deviceProp.maxThreadsDim[1];

    // int maxThreadPB = deviceProp.maxThreadsPerBlock;

    *block = dim3(xThreads, yThreads, 1);

    int xBlocks = (int) std::ceil(((double)width) / xThreads);
    int yBlocks = (int) std::ceil(((double)height) / yThreads);
    *grid = dim3(xBlocks, yBlocks, 1);
}


__global__ void compute_distance(double *m, size_t m_p, double *p, size_t p_p,
                                 double *distance, size_t distance_p,  int xSize, int ySize){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i >= xSize || j >= ySize)
        return;

    m_p = m_p/sizeof(double);
    double mx = m[i];
    double my = m[i + m_p];
    double mz = m[i + 2*m_p];

    p_p = p_p/sizeof(double);
    double x = p[j] - mx;
    double y = p[j + p_p] - my;
    double z = p[j + 2*p_p] - mz;

    distance_p = distance_p/sizeof(double);
    distance[i + j*distance_p] = x*x + y*y + z*z;

}



__global__ void find_Y(double *distance, size_t distance_p,
                       double *m, size_t m_p, double *Y, size_t Y_p,
                       int xSize, int ySize) {

    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (j >= ySize)
        return;

    distance_p = distance_p/sizeof(double);
    Y_p = Y_p / sizeof(double);
    m_p = m_p/ sizeof(double);

    int minIdx = 0;
    for (int i = 1; i < xSize; i++) {
        if (distance[minIdx + j*distance_p] > distance[i + j*distance_p]){
            minIdx = i;
        }
    }

    double mx = m[minIdx];
    double my = m[minIdx + m_p];
    double mz = m[minIdx + 2*m_p];

    Y[j] = mx;
    Y[j+ Y_p] = my;
    Y[j+ 2*Y_p] = mz;
}

void compute_Y_w(const GPU::Matrix &m, const GPU::Matrix &p, GPU::Matrix &Y){
    size_t m_p, p_p, Y_p;
    double *m_gpu = m.toGpu(&m_p);
    double *p_gpu = p.toGpu(&p_p);
    double *Y_gpu = Y.toGpu(&Y_p);


    double *distance;
    size_t distance_p;
    hipMallocPitch((void **) &distance, &distance_p, sizeof(double) * m.cols(), p.cols());

    dim3 distBlk, distGrd;
    computeDim(m.cols(), p.cols(), &distBlk, &distGrd);
    compute_distance<<<distGrd, distBlk>>>(m_gpu, m_p, p_gpu, p_p, distance, distance_p, m.cols(), p.cols());
    hipDeviceSynchronize();

    hipFree(p_gpu);

    dim3 YBlk, YGrd;
    YBlk = dim3(1, 32, 1);
    int xBlocks = 1;
    int yBlocks = (int) std::ceil(((double) p.cols()) / 32);
    YGrd = dim3(xBlocks, yBlocks, 1);
    find_Y<<<YGrd, YBlk>>>(distance, distance_p, m_gpu, m_p, Y_gpu, Y_p, m.cols(), p.cols());
    hipDeviceSynchronize();

    hipFree(m_gpu);
    hipFree(distance);

    Y.fromGpu(Y_gpu, Y.rows(), Y.cols(), Y_p);

    hipFree(Y_gpu);
}

__global__ void compute_err(double *Y_gpu, double *p_gpu, double *sr_gpu, double
                            *t_gpu, double *err, size_t Y_p, size_t p_p, size_t sr_p,
                            size_t t_p, size_t err_p, unsigned int size)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    Y_p = Y_p / sizeof(double);
    p_p = p_p / sizeof(double);
    sr_p = sr_p / sizeof(double);
    t_p = t_p / sizeof(double);
    err_p = err_p / sizeof(double);

    double px = sr_gpu[0] * p_gpu[i] + sr_gpu[1] * p_gpu[i + p_p] + sr_gpu[2] * p_gpu[i + 2* p_p];
    double py = sr_gpu[sr_p] * p_gpu[i] + sr_gpu[1 + sr_p] * p_gpu[i + p_p] + sr_gpu[2 + sr_p] * p_gpu[i + 2* p_p];
    double pz = sr_gpu[2 * sr_p] * p_gpu[i] + sr_gpu[1 + 2*sr_p] * p_gpu[i + p_p] + sr_gpu[2 + 2*sr_p] * p_gpu[i + \
                                                                                                               2 * p_p];

    p_gpu[i] = px + t_gpu[0];
    p_gpu[i + p_p] = py + t_gpu[t_p];
    p_gpu[i + 2*p_p] = pz + t_gpu[2*t_p];

    Y_gpu[i] = Y_gpu[i] - p_gpu[i];
    Y_gpu[i + Y_p] = Y_gpu[i + Y_p] - p_gpu[i + p_p];
    Y_gpu[i + 2*Y_p] = Y_gpu[i + 2*Y_p] - p_gpu[i + 2*p_p];
    err[i] = Y_gpu[i] * Y_gpu[i] + Y_gpu[i + Y_p] * Y_gpu[i + Y_p] + Y_gpu[i + 2*Y_p] * Y_gpu[i + 2*Y_p];
}


double compute_err_w(const GPU::Matrix &Y, GPU::Matrix &p, bool in_place,
                     const GPU::Matrix &sr, const GPU::Matrix &t)
{
    size_t p_p, sr_p, t_p, Y_p;
    double *p_gpu = p.toGpu(&p_p);
    double *sr_gpu = sr.toGpu(&sr_p);
    double *t_gpu = t.toGpu(&t_p);
    double *Y_gpu =Y.toGpu(&Y_p);

    size_t err_p;
    GPU::Matrix tmp {MatrixXd{1,Y.cols()}};
    double *err = tmp.toGpu(&err_p);

    dim3 PBlk, PGrd;
    PBlk = dim3(32, 1, 1);
    int xBlocks = (int) std::ceil(((double) p.cols()) / 32);
    int yBlocks = 1;
    PGrd = dim3(xBlocks, yBlocks, 1);
    compute_err<<<PGrd,PBlk>>>(Y_gpu, p_gpu, sr_gpu, t_gpu, err, Y_p, p_p,
                               sr_p, t_p, err_p, p.cols());
    hipDeviceSynchronize();

    if (!in_place)
        p.fromGpu(p_gpu, p.rows(), p.cols(), p_p);

    hipFree(p_gpu);
    hipFree(sr_gpu);
    hipFree(t_gpu);

    tmp.fromGpu(err, 1, tmp.cols(), err_p);
    hipFree(Y_gpu);
    hipFree(err);

    return tmp.sum();
}
